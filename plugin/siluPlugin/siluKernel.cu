#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#if CUDA_VERSION >= 10010

#include <cassert>
#include <cstring>
#include <vector>

#include "siluPlugin.h"

using namespace nvinfer1;

__device__ __forceinline__ float sigmoid (float a)
{
    return 1.0 / (1.0 + exp (-a));
}

void __global__ silu_kernel(int n, float* inputData, float* outputData) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        inputData[tid] = outputData[tid] * sigmoid(outputData[tid]);
    }
}

int SiLUInference(hipStream_t stream, int size, float* inputData, float* outputData){
    const int nThreads = 512;

    int nBlocks = (size + nThreads - 1) / nThreads;
    
    silu_kernel<<<nThreads, nBlocks>>>(size, inputData, outputData);

    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
             __FILE__, __LINE__, hipGetErrorString( err ) );
        return 1;
    }
    return 0;
}

#endif // CUDA_VERSION >= 10010
